#include "hip/hip_runtime.h"
#include<cstdio.h>

int main(int argc, char const *argv[]) {
  int N = argv[1];
  int a = argv[2];
  int b = argv[3];
  int tab_CPU[N];

  for (int i=0; i<N; i++){
    tab_CPU[i] = i;
  }

  // Allocate vector in device memory
  hipMalloc(&tab_GPU, N * sizeof(int));
  // Copy vectors from host memory to device memory
  hipMemcpy(tab_GPU, tab_CPU, N * sizeof(int), hipMemcpyHostToDevice);

  saxpy<<<1,N>>>(tab_GPU, N, a, b);

  hipMemcpy(tab_CPU, tab_GPU, N * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(tab_GPU);

  return 0;
}


__global__ void saxpy(int *tab, int N, int a, int b){
  int idx = threadIdx.x;
  tab[idx+1] = a * tab[idx+1] + b;
}
