
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void saxpy(int *tab, int N, int a, int b);

int main(int argc, char const *argv[]) {
  int N = atoi(argv[1]); 
  int a = atoi(argv[2]); 
  int b = atoi(argv[3]);
  int N_threads = atoi(argv[4]);

  int tab_CPU[N];
  for (int i=0; i<N; i++){
    tab_CPU[i] = i;
  }

  int *tab_GPU;
  // Allocate vector in device memory
  hipMalloc(&tab_GPU, N * sizeof(int));
  // Copy vectors from host memory to device memory
  hipMemcpy(tab_GPU, tab_CPU, N * sizeof(int), hipMemcpyHostToDevice);

  int threadsPerBlock = N_threads;
  int blocksPerGrid =
            (int) ceil(N / (float)threadsPerBlock);

  saxpy<<<blocksPerGrid,threadsPerBlock>>>(tab_GPU, N, a, b);

  hipMemcpy(tab_CPU, tab_GPU, N * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(tab_GPU);

  for (int i=0; i<20; i++){
    printf("%d ",tab_CPU[i]);
  }

  return 0;
}


__global__ void saxpy(int *tab, int N, int a, int b){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx<=N)
    tab[idx] = a * tab[idx] + b;
}
